#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "defConstants.cuh"
#include "genData.cuh"
#include "convecKernels.cuh"

//__constant__ float xgp_cte[MAX_NGAUS];
__constant__ float wgp_cte[MAX_NGAUS];
//__constant__ float N_cte[MAX_NGAUS*MAX_NNODE];
//__constant__ float dN_cte[MAX_NGAUS*MAX_NNODE];

__global__ void convec_gpuConst(int nelem, int nnode, int ngaus, int npoints, int *connec,
                                float *N, float *dN, float *w, float *u, float *R)
{
    // Create shared memory
    __shared__ float u_shared[8]; // Max 3 nodes per element
    __shared__ float v_shared[8]; // Max 3 Gauss points per element

    // Set ielem and inode according to block and thread indices
    int ielem = blockIdx.x;
    int inode = threadIdx.x;
    int igaus = threadIdx.y;

    // Ensure R is zero
    R[connec[ielem*nnode + inode]] = 0.0f;

    // Fill shared memory
    v_shared[igaus] = 0.0f;
    u_shared[inode] = u[connec[ielem*nnode + inode]];
    __syncthreads();

    // Compute dN*u_shared at each Gauss point
    //for (int jnode = 0; jnode < nnode; jnode++)
    //{
    //	v_shared[igaus] += dN[igaus*nnode + jnode]*u_shared[jnode];
    //}
    //__syncthreads();
    atomicAdd(&v_shared[igaus], dN[igaus*nnode + inode]*u_shared[inode]);

    // Atomically update R
    atomicAdd(&R[connec[ielem*nnode + inode]], wgp_cte[igaus]*N[igaus*nnode + inode]*v_shared[igaus]);
    __syncthreads();
}

int main(void)
{
    // Var for error checking
    int ierr;

    // Set mesh details
    int nelem = 2;
    int nnode = 3;
    int ngaus = 3;
    int npoints = 5;

    // Print mesh info
    printf("*----------*\n");
    printf("nelem = %d\n", nelem);
    printf("nnode = %d\n", nnode);
    printf("ngaus = %d\n", ngaus);
    printf("npoints = %d\n", npoints);
    printf("*----------*\n");

    // Create connectivity table
    int *connec = (int *)malloc(nelem*nnode*sizeof(int));
    connec[0] = 0;
    connec[1] = 1;
    connec[2] = 3;
    connec[3] = 1;
    connec[4] = 2;
    connec[5] = 4;

    // Print connec in a table format
    printf("connec = \n");
    for (int ielem = 0; ielem < nelem; ielem++)
    {
        for (int inode = 0; inode < nnode; inode++)
        {
            printf("%d ", connec[ielem*nnode + inode]);
        }
        printf("\n");
    }
    printf("*----------*\n");

    // Set quadrature points
    float *xgp = (float *)malloc(ngaus*sizeof(float));
    float *wgp = (float *)malloc(ngaus*sizeof(float));
    ierr = quadratureData(ngaus,xgp,wgp);
    if (ierr != 0)
    {
        printf("Error in quadratureData\n");
        return EXIT_FAILURE;
    }
    //wgp[0] = 1.0f;
    //wgp[1] = 1.0f;
    //wgp[2] = 1.0f;

    // Set N and dN
    float *N = (float *)malloc(nnode*ngaus*sizeof(float));
    float *dN = (float *)malloc(nnode*ngaus*sizeof(float));
    for (int igaus = 0; igaus < ngaus; igaus++)
    {
        for (int inode = 0; inode < nnode; inode++)
        {
            N[igaus*nnode + inode] = 1.0f;
            dN[igaus*nnode + inode] = 0.5f;
        }
    }

    // Set initial condition u
    float *u = (float *)malloc(npoints*sizeof(float));
    u[0] = 1.0f;
    u[1] = 2.0f;
    u[2] = 1.0f;
    u[3] = 1.5f;
    u[4] = 1.5f;

    // Print u
    printf("u = \n");
    for (int ipoint = 0; ipoint < npoints; ipoint++)
    {
        printf("%d %f\n", ipoint, u[ipoint]);
    }
    printf("*----------*\n");

    // Call the CPU version of convec
    float *R_cpu = (float *)malloc(npoints*sizeof(float));
    convec_cpu(nelem,nnode,ngaus,npoints,connec,
                N,dN,wgp,u,R_cpu);

    // Print R_cpu
    printf("R_cpu = \n");
    for (int ipoint = 0; ipoint < npoints; ipoint++)
    {
        printf("%d %f\n", ipoint, R_cpu[ipoint]);
    }
    printf("*----------*\n");

    // Create GPU arrays for connec, N, dN, wgp, u, and R_gpu
    int *connec_gpu;
    float *N_gpu, *dN_gpu, *wgp_gpu, *u_gpu, *R_gpu;

    hipMalloc((void **)&connec_gpu, nelem*nnode*sizeof(int));
    hipMalloc((void **)&N_gpu, nnode*ngaus*sizeof(float));
    hipMalloc((void **)&dN_gpu, nnode*ngaus*sizeof(float));
    hipMalloc((void **)&wgp_gpu, ngaus*sizeof(float));
    hipMalloc((void **)&u_gpu, npoints*sizeof(float));
    hipMalloc((void **)&R_gpu, npoints*sizeof(float));

    hipMemcpy(connec_gpu, connec, nelem*nnode*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(N_gpu, N, nnode*ngaus*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dN_gpu, dN, nnode*ngaus*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(wgp_gpu, wgp, ngaus*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(u_gpu, u, npoints*sizeof(float), hipMemcpyHostToDevice);

    // Call the basic GPU version of convec
    convec_gpuBasic<<<nelem,nnode>>>(nelem,nnode,ngaus,npoints,connec_gpu,
                                     N_gpu,dN_gpu,wgp_gpu,u_gpu,R_gpu);
    
    // Copy data from GPU to CPU
    float *R_gpuBasic = (float *)malloc(npoints*sizeof(float));
    hipMemcpy(R_gpuBasic, R_gpu, npoints*sizeof(float), hipMemcpyDeviceToHost);

    // Print R_gpuBasic
    printf("R_gpuBasic = \n");
    for (int ipoint = 0; ipoint < npoints; ipoint++)
    {
        printf("%d %f\n", ipoint, R_gpuBasic[ipoint]);
    }
    printf("*----------*\n");

    // Call the 1st shared memory GPU version of convec
    convec_gpuShared1<<<nelem,nnode>>>(nelem,nnode,ngaus,npoints,connec_gpu,
                                       N_gpu,dN_gpu,wgp_gpu,u_gpu,R_gpu);

    // Copy data from GPU to CPU
    float *R_gpuShared1 = (float *)malloc(npoints*sizeof(float));
    hipMemcpy(R_gpuShared1, R_gpu, npoints*sizeof(float), hipMemcpyDeviceToHost);

    // Print R_gpuShared1
    printf("R_gpuShared1 = \n");
    for (int ipoint = 0; ipoint < npoints; ipoint++)
    {
        printf("%d %f\n", ipoint, R_gpuShared1[ipoint]);
    }
    printf("*----------*\n");

    // Call the 2nd shared memory GPU version of convec
    dim3 block(nnode,ngaus,1);
    dim3 grid(nelem,1,1);
    convec_gpuShared2<<<grid,block>>>(nelem,nnode,ngaus,npoints,connec_gpu,
                                      N_gpu,dN_gpu,wgp_gpu,u_gpu,R_gpu);

    // Copy data from GPU to CPU
    float *R_gpuShared2 = (float *)malloc(npoints*sizeof(float));
    hipMemcpy(R_gpuShared2, R_gpu, npoints*sizeof(float), hipMemcpyDeviceToHost);

    // Print R_gpuShared2
    printf("R_gpuShared2 = \n");
    for (int ipoint = 0; ipoint < npoints; ipoint++)
    {
        printf("%d %f\n", ipoint, R_gpuShared2[ipoint]);
    }
    printf("*----------*\n");

    // Fill the constant memory wgp_cte
    hipMemcpyToSymbol(HIP_SYMBOL(wgp_cte), wgp, ngaus*sizeof(float), 0, hipMemcpyHostToDevice);
    
    // Call the constant memory GPU version of convec
    convec_gpuConst<<<grid,block>>>(nelem,nnode,ngaus,npoints,connec_gpu,
                                     N_gpu,dN_gpu,wgp_gpu,u_gpu,R_gpu);

    // Copy data from GPU to CPU
    float *R_gpuConst = (float *)malloc(npoints*sizeof(float));
    hipMemcpy(R_gpuConst, R_gpu, npoints*sizeof(float), hipMemcpyDeviceToHost);

    // Print R_gpuConst
    printf("R_gpuConst = \n");
    for (int ipoint = 0; ipoint < npoints; ipoint++)
    {
        printf("%d %f\n", ipoint, R_gpuConst[ipoint]);
    }
    printf("*----------*\n");

    return 0;
}